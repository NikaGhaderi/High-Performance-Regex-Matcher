// gpu_matcher.cu
// Implements a memory-efficient, two-pass iterative regex matcher using
// Thompson's NFA construction to avoid recursion and handle large datasets.


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <chrono>
#include <numeric>
#include <sys/stat.h>
#include <stack>

// ==================================================================================
// CUDA ERROR CHECKING MACRO
// ==================================================================================
#define CUDA_CHECK(call)                                                    \
do {                                                                        \
    hipError_t err = call;                                                 \
    if (err != hipSuccess) {                                               \
        fprintf(stderr, "CUDA Error in %s at line %d: %s\n",                \
                __FILE__, __LINE__, hipGetErrorString(err));               \
        exit(EXIT_FAILURE);                                                 \
    }                                                                       \
} while (0)

// ==================================================================================
// NFA DATA STRUCTURES
// ==================================================================================
struct Match { int line_id; int pattern_id; };
enum StateType { MATCH = 256, SPLIT = 257 };
struct NfaState { int c; int next1; int next2; };
struct NfaFragment { int start_node; int end_node; };

// ==================================================================================
// NFA COMPILER (HOST-SIDE)
// ==================================================================================
int compile_pattern_to_nfa(const std::string& pattern, std::vector<NfaState>& nfa_states) {
    std::stack<NfaFragment> fragment_stack;
    int initial_nfa_size = nfa_states.size();

    auto add_state = [&](int c, int n1, int n2) {
        nfa_states.push_back({c, n1, n2});
        return (int)nfa_states.size() - 1;
    };

    for (char p_char : pattern) {
        if (p_char == '*') {
            if (fragment_stack.empty()) continue;
            NfaFragment frag = fragment_stack.top(); fragment_stack.pop();
            int split_state = add_state(SPLIT, frag.start_node, -1);
            nfa_states[frag.end_node].c = SPLIT;
            nfa_states[frag.end_node].next1 = frag.start_node;
            nfa_states[frag.end_node].next2 = split_state;
            fragment_stack.push({split_state, split_state});
        } else {
            int s = add_state(p_char, -1, -1);
            fragment_stack.push({s, s});
        }
    }

    NfaFragment final_frag = {-1, -1};
    if (!fragment_stack.empty()) {
        final_frag = fragment_stack.top(); fragment_stack.pop();
        while(!fragment_stack.empty()){
            NfaFragment prev_frag = fragment_stack.top(); fragment_stack.pop();
            nfa_states[prev_frag.end_node].next1 = final_frag.start_node;
            final_frag.start_node = prev_frag.start_node;
        }
    }

    int match_state = add_state(MATCH, -1, -1);
    if (final_frag.start_node != -1) {
        nfa_states[final_frag.end_node].next1 = match_state;
        for (size_t i = initial_nfa_size; i < nfa_states.size(); ++i) {
            if (nfa_states[i].c == SPLIT && nfa_states[i].next2 == -1) {
                nfa_states[i].next2 = match_state;
            }
        }
        return final_frag.start_node;
    }
    return match_state;
}

// ==================================================================================
// GPU KERNEL AND DEVICE FUNCTIONS
// ==================================================================================
__device__ void add_state_with_closure(int* state_list, int& count, int state_idx, const NfaState* nfa_states, int max_states) {
    if (state_idx == -1 || count >= max_states) return;
    int stack[32]; int stack_ptr = 0; stack[stack_ptr++] = state_idx;
    while (stack_ptr > 0) {
        int current_state_idx = stack[--stack_ptr];
        bool found = false;
        for (int i = 0; i < count; ++i) if (state_list[i] == current_state_idx) { found = true; break; }
        if (found) continue;
        state_list[count++] = current_state_idx;
        if (nfa_states[current_state_idx].c == SPLIT) {
            if (nfa_states[current_state_idx].next1 != -1 && stack_ptr < 32) stack[stack_ptr++] = nfa_states[current_state_idx].next1;
            if (nfa_states[current_state_idx].next2 != -1 && stack_ptr < 32) stack[stack_ptr++] = nfa_states[current_state_idx].next2;
        }
    }
}

__device__ bool nfa_match(const NfaState* nfa_states, int start_state, const char* text, int text_len) {
    const int MAX_ACTIVE_STATES = 128;
    int current_states[MAX_ACTIVE_STATES], next_states[MAX_ACTIVE_STATES];
    int current_count = 0, next_count = 0;
    add_state_with_closure(current_states, current_count, start_state, nfa_states, MAX_ACTIVE_STATES);
    for (int i = 0; i < text_len; ++i) {
        char c = text[i]; next_count = 0;
        for (int j = 0; j < current_count; ++j) {
            int state_idx = current_states[j];
            const NfaState& state = nfa_states[state_idx];
            if (state.c == c || state.c == '.') {
                add_state_with_closure(next_states, next_count, state.next1, nfa_states, MAX_ACTIVE_STATES);
            }
        }
        for(int k=0; k<next_count; ++k) current_states[k] = next_states[k];
        current_count = next_count;
    }
    for (int i = 0; i < current_count; ++i) {
        if (nfa_states[current_states[i]].c == MATCH) return true;
    }
    return false;
}

/**
 * @brief KERNEL PASS 1: Counts total matches without storing them.
 */
__global__ void regex_count_kernel(
    const char* d_lines_flat, const int* d_line_offsets, const int* d_line_lengths, int num_lines,
    const NfaState* d_nfa_states, const int* d_pattern_start_states, int num_patterns,
    int* d_match_count) {

    int line_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (line_idx >= num_lines) return;

    const char* line = d_lines_flat + d_line_offsets[line_idx];
    int line_len = d_line_lengths[line_idx];

    for (int pattern_idx = 0; pattern_idx < num_patterns; ++pattern_idx) {
        int start_state = d_pattern_start_states[pattern_idx];
        for (int i = 0; i < line_len; ++i) {
            if (nfa_match(d_nfa_states, start_state, line + i, line_len - i)) {
                atomicAdd(d_match_count, 1);
                break; 
            }
        }
    }
}

/**
 * @brief KERNEL PASS 2: Stores match results into a pre-sized buffer.
 */
__global__ void regex_store_kernel(
    const char* d_lines_flat, const int* d_line_offsets, const int* d_line_lengths, int num_lines,
    const NfaState* d_nfa_states, const int* d_pattern_start_states, int num_patterns,
    Match* d_results, int* d_match_count) {

    int line_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (line_idx >= num_lines) return;

    const char* line = d_lines_flat + d_line_offsets[line_idx];
    int line_len = d_line_lengths[line_idx];

    for (int pattern_idx = 0; pattern_idx < num_patterns; ++pattern_idx) {
        int start_state = d_pattern_start_states[pattern_idx];
        for (int i = 0; i < line_len; ++i) {
            if (nfa_match(d_nfa_states, start_state, line + i, line_len - i)) {
                int match_idx = atomicAdd(d_match_count, 1);
                d_results[match_idx].line_id = line_idx;
                d_results[match_idx].pattern_id = pattern_idx;
                break;
            }
        }
    }
}

// ==================================================================================
// HOST HELPER FUNCTIONS (Unchanged)
// ==================================================================================
std::vector<std::string> read_lines(const std::string& filename, size_t& total_bytes) {
    std::vector<std::string> lines;
    std::ifstream file(filename);
    if (!file.is_open()) { std::cerr << "Error: Could not open file " << filename << std::endl; exit(EXIT_FAILURE); }
    std::string line;
    total_bytes = 0;
    while (std::getline(file, line)) {
        if (!line.empty() && (line.back() == '\r' || line.back() == '\n')) line.pop_back();
        lines.push_back(line);
        total_bytes += line.length();
    }
    return lines;
}

void flatten_strings(const std::vector<std::string>& strings, std::vector<char>& flat, std::vector<int>& offsets, std::vector<int>& lengths) {
    int current_offset = 0;
    for (const auto& s : strings) {
        flat.insert(flat.end(), s.begin(), s.end());
        flat.push_back('\0');
        offsets.push_back(current_offset);
        lengths.push_back(s.length());
        current_offset += s.length() + 1;
    }
}

void write_metrics(const std::string& filename, const std::string& matcher_name, double throughput_input, double throughput_mbytes, double throughput_matches, double latency) {
    std::ofstream file(filename, std::ios_base::app);
    if (!file.is_open()) { std::cerr << "Error: Could not open metrics file " << filename << std::endl; return; }
    file.seekp(0, std::ios::end);
    if (file.tellp() == 0) file << "matcher_name,throughput_input_per_sec,throughput_mbytes_per_sec,throughput_match_per_sec,latency\n";
    file << matcher_name << "," << std::fixed << throughput_input << "," << throughput_mbytes << "," << throughput_matches << "," << latency << "\n";
}

void write_output(const std::string& filename, const std::vector<Match>& matches, int num_lines) {
    std::ofstream file(filename);
    if (!file.is_open()) { std::cerr << "Error: Could not open output file " << filename << std::endl; return; }
    std::vector<std::vector<int>> grouped_matches(num_lines);
    for (const auto& match : matches) {
        if(match.line_id < num_lines) grouped_matches[match.line_id].push_back(match.pattern_id);
    }
    for (int i = 0; i < num_lines; ++i) {
        for (size_t j = 0; j < grouped_matches[i].size(); ++j) {
            file << grouped_matches[i][j] << (j == grouped_matches[i].size() - 1 ? "" : ",");
        }
        file << "\n";
    }
}

// ==================================================================================
// MAIN FUNCTION (Modified for Two-Pass Execution)
// ==================================================================================
int main(int argc, char* argv[]) {
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <patterns_file> <input_file> <output_file> <metrics_file>" << std::endl;
        return 1;
    }

    std::string patterns_file = argv[1], input_file = argv[2], output_file = argv[3], metrics_file = argv[4];

    // --- 1. Read and Compile Data on Host ---
    std::cout << "Reading input files..." << std::endl;
    size_t pattern_bytes, input_bytes;
    std::vector<std::string> h_patterns = read_lines(patterns_file, pattern_bytes);
    std::vector<std::string> h_lines = read_lines(input_file, input_bytes);
    std::cout << "Read " << h_patterns.size() << " patterns and " << h_lines.size() << " input lines." << std::endl;

    std::cout << "Compiling " << h_patterns.size() << " patterns to NFA graph..." << std::endl;
    std::vector<NfaState> h_nfa_states;
    std::vector<int> h_pattern_start_states;
    for (const auto& pattern : h_patterns) {
        h_pattern_start_states.push_back(compile_pattern_to_nfa(pattern, h_nfa_states));
    }
    std::cout << "NFA compilation complete. Total states: " << h_nfa_states.size() << std::endl;

    std::vector<char> h_lines_flat;
    std::vector<int> h_line_offsets, h_line_lengths;
    flatten_strings(h_lines, h_lines_flat, h_line_offsets, h_line_lengths);

    // --- 2. Allocate and Transfer Common Data to GPU ---
    std::cout << "Allocating GPU memory and transferring common data..." << std::endl;
    char *d_lines_flat; int *d_line_offsets, *d_line_lengths;
    NfaState* d_nfa_states; int* d_pattern_start_states; int* d_match_count;
    CUDA_CHECK(hipMalloc(&d_lines_flat, h_lines_flat.size()));
    CUDA_CHECK(hipMalloc(&d_line_offsets, h_line_offsets.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_line_lengths, h_line_lengths.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_nfa_states, h_nfa_states.size() * sizeof(NfaState)));
    CUDA_CHECK(hipMalloc(&d_pattern_start_states, h_pattern_start_states.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_match_count, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_lines_flat, h_lines_flat.data(), h_lines_flat.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_line_offsets, h_line_offsets.data(), h_line_offsets.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_line_lengths, h_line_lengths.data(), h_line_lengths.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_nfa_states, h_nfa_states.data(), h_nfa_states.size() * sizeof(NfaState), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pattern_start_states, h_pattern_start_states.data(), h_pattern_start_states.size() * sizeof(int), hipMemcpyHostToDevice));
    
    // --- 3. PASS 1: Execute Counting Kernel ---
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start)); CUDA_CHECK(hipEventCreate(&stop));
    std::cout << "Launching CUDA kernel (Pass 1: Counting)..." << std::endl;
    CUDA_CHECK(hipMemset(d_match_count, 0, sizeof(int)));

    CUDA_CHECK(hipEventRecord(start));
    int threads_per_block = 256;
    int num_blocks = (h_lines.size() + threads_per_block - 1) / threads_per_block;
    regex_count_kernel<<<num_blocks, threads_per_block>>>(
        d_lines_flat, d_line_offsets, d_line_lengths, h_lines.size(),
        d_nfa_states, d_pattern_start_states, h_patterns.size(),
        d_match_count
    );
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float pass1_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&pass1_ms, start, stop));

    int h_match_count = 0;
    CUDA_CHECK(hipMemcpy(&h_match_count, d_match_count, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "Pass 1 complete. Found " << h_match_count << " total matches." << std::endl;

    // --- 4. Allocate Exact Memory for Results ---
    Match* d_results;
    if (h_match_count > 0) {
        std::cout << "Allocating exact memory for results..." << std::endl;
        CUDA_CHECK(hipMalloc(&d_results, h_match_count * sizeof(Match)));
    }

    // --- 5. PASS 2: Execute Storing Kernel ---
    std::cout << "Launching CUDA kernel (Pass 2: Storing)..." << std::endl;
    CUDA_CHECK(hipMemset(d_match_count, 0, sizeof(int))); // Reset counter for storing

    CUDA_CHECK(hipEventRecord(start));
    regex_store_kernel<<<num_blocks, threads_per_block>>>(
        d_lines_flat, d_line_offsets, d_line_lengths, h_lines.size(),
        d_nfa_states, d_pattern_start_states, h_patterns.size(),
        d_results, d_match_count
    );
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float pass2_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&pass2_ms, start, stop));
    double total_time_sec = (pass1_ms + pass2_ms) / 1000.0;
    std::cout << "Kernel execution took " << total_time_sec << " seconds (Pass 1: " << pass1_ms/1000.0 << "s, Pass 2: " << pass2_ms/1000.0 << "s)." << std::endl;

    // --- 6. Transfer Results Back to Host ---
    std::cout << "Transferring results back to CPU..." << std::endl;
    std::vector<Match> h_results;
    if (h_match_count > 0) {
        h_results.resize(h_match_count);
        CUDA_CHECK(hipMemcpy(h_results.data(), d_results, h_match_count * sizeof(Match), hipMemcpyDeviceToHost));
    }

    // --- 7. Calculate Metrics and Write Output ---
    std::cout << "Processing results and writing output files..." << std::endl;
    double throughput_input = (total_time_sec > 0) ? h_lines.size() / total_time_sec : 0;
    double throughput_mbytes = (total_time_sec > 0) ? (input_bytes / (1024.0 * 1024.0)) / total_time_sec : 0;
    double throughput_matches = (total_time_sec > 0) ? h_match_count / total_time_sec : 0;
    double latency = (h_lines.size() > 0) ? (total_time_sec * 1000.0) / h_lines.size() : 0;

    write_output(output_file, h_results, h_lines.size());
    write_metrics(metrics_file, "CustomGPU_NFA_TwoPass", throughput_input, throughput_mbytes, throughput_matches, latency);

    // --- 8. Cleanup ---
    std::cout << "Cleaning up GPU memory." << std::endl;
    CUDA_CHECK(hipEventDestroy(start)); CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_lines_flat)); CUDA_CHECK(hipFree(d_line_offsets)); CUDA_CHECK(hipFree(d_line_lengths));
    CUDA_CHECK(hipFree(d_nfa_states)); CUDA_CHECK(hipFree(d_pattern_start_states));
    if (h_match_count > 0) CUDA_CHECK(hipFree(d_results));
    CUDA_CHECK(hipFree(d_match_count));

    std::cout << "Processing complete." << std::endl;
    printf("  - Total Matches:          %d\n", h_match_count);
    printf("  - Throughput (input/sec): %.2f\n", throughput_input);
    printf("  - Throughput (MB/sec):    %.2f\n", throughput_mbytes);
    printf("  - Throughput (match/sec): %.2f\n", throughput_matches);
    printf("  - Latency (ms/line):      %.4f\n", latency);

    return 0;
}
